#include "hip/hip_runtime.h"
#include <exception>
#include <list>
#include "util/macro.hh"
#include "pump/flat.hh"
#include "pump/repeat.hh"
#include "pump/sequential.hh"
#include "pump/reduce.hh"
#include "pump/visit.hh"
#include "pump/just.hh"
#include "pump/when_all.hh"
#include "pump/submit.hh"
#include "sider/coro/coro.hh"
#include "sider/cuda/common/wait_stream_done.hh"

#include "hip/hip_runtime.h"


using namespace pump::coro;
using namespace sider::pump;
using namespace sider::meta;
using namespace sider::cuda;

#define N 10000000
template <typename T>
__global__
void vector_add(T *out, T *a, T *b, int n) {

    for(int i = 0; i < n; i++){
        out[i] = a[i] * b[i];
    }
    printf("1111  %d,%d\n", (int) out[0], (int) out[1]);
}

struct
compute_unit {
    float *g_a, *g_b, *g_o;
    hipStream_t stream;
};
int
main(int argc, char **argv) {


    std::list<compute_unit*> units;
    just()
        >> for_each(units)
        >> then([](compute_unit* u){

        })
        >> reduce();

    float *c_a, *c_b, *c_o;
    float *g_a, *g_b, *g_o;

    hipStream_t stream{0};
    hipStreamCreate(&stream);

    just()
        >> then([&]() {
            c_a   = (float*)malloc(sizeof(float) * N);
            hipMalloc((void **) &g_a, sizeof(float) * N);

            hipPointerAttribute_t attr;
            auto e = hipPointerGetAttributes(&attr, g_a);

            memcpy(attr.devicePointer, c_a, sizeof(float) * N);

            c_b   = (float*)malloc(sizeof(float) * N);
            hipMalloc((void **) &g_b, sizeof(float) * N);
            c_o   = (float*)malloc(sizeof(float) * N);
            hipMalloc((void **) &g_o, sizeof(float) * N);

            for(int i = 0; i < N; i++){
                c_a[i] = 1.0f; c_b[i] = 2.0f;
            }

            hipMemcpyAsync(g_a, c_a, sizeof(float) * N, hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(g_b, c_b, sizeof(float) * N, hipMemcpyHostToDevice, stream);
            vector_add<<<1, 1, 0, stream>>>(g_o, g_a, g_b, N);
            hipMemcpyAsync(c_o, g_o, sizeof(float) * N, hipMemcpyDeviceToHost, stream);
        })
        >> common::wait_stream_done(stream)
        >> then([](){
            std::cout << "done" << std::endl;
        })
        >> submit(make_root_context());

    sleep(100);
    return 0;
}